#include "hip/hip_runtime.h"
﻿#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""

#define N 255

__global__ void print(void) {
    int i = blockIdx.x;
    printf("%d", &i);
}

int main()
{
    print << <1, 1 >> > ();

}
