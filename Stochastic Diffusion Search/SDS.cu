
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const int N = 10;
const int maxIter = 30;

__device__ float r(hiprandState* state) {
    // Generate a random number in the range [0, 1] using CURAND
    return hiprand_uniform(state);
}

__global__ void sdsKernel(char* ss, char* model, int ssLength, int modelLength, int* hypo, bool* status) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(clock64(), tid, 0, &state); // Initialize the random state for this thread

    if (tid < N) {
        int hypoValue = hypo[tid];
        bool agentStatus = false;

        for (int itr = 0; itr < maxIter; itr++) {
            int microFeature = r(&state) * modelLength;

            if (ss[hypoValue + microFeature] == model[microFeature]) {
                agentStatus = true;
            } else {
                agentStatus = false;
            }

            __syncthreads();

            if (!agentStatus) {
                int randAgent = r(&state) * N;
                if (status[randAgent]) {
                    hypoValue = hypo[randAgent];
                } else {
                    hypoValue = r(&state) * (ssLength - modelLength);
                }
            } else {
                microFeature = r(&state) * modelLength;
                if (ss[hypoValue + microFeature] == model[microFeature]) {
                    agentStatus = true;
                } else {
                    agentStatus = false;
                }
            }

            __syncthreads();
        }
        hypo[tid] = hypoValue;
        status[tid] = agentStatus;
    }
}

int main() {
    string ss = "try to find sds in this sentence";
    string model = " sds ";

    int ssLength = ss.length();
    int modelLength = model.length();

    char* d_ss;
    char* d_model;
    int* d_hypo;
    bool* d_status;

    // Allocate device memory
    hipMalloc((void**)&d_ss, ssLength);
    hipMalloc((void**)&d_model, modelLength);
    hipMalloc((void**)&d_hypo, N * sizeof(int));
    hipMalloc((void**)&d_status, N * sizeof(bool));

    // Copy data from host to device
    hipMemcpy(d_ss, ss.c_str(), ssLength, hipMemcpyHostToDevice);
    hipMemcpy(d_model, model.c_str(), modelLength, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    sdsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_ss, d_model, ssLength, modelLength, d_hypo, d_status);

    // Copy results from device to host
    int* hypo = new int[N];
    bool* status = new bool[N];
    hipMemcpy(hypo, d_hypo, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(status, d_status, N * sizeof(bool), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_ss);
    hipFree(d_model);
    hipFree(d_hypo);
    hipFree(d_status);

    for (int i = 0; i < N; i++) {
        cout << "Agent " << i << " - Found: " << ss.substr(hypo[i], modelLength) << " Status: " << status[i] << endl;
    }

    return 0;
}
