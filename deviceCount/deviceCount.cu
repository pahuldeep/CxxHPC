﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
using namespace std;

int main() {
	int device_count = 0;

	if (device_count == hipGetDevice(&device_count)) printf("No device");
	else printf("device avail");

	printf("%d", hipGetDevice(&device_count));

	hipDeviceSynchronize();

}
