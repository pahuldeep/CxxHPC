#include "hip/hip_runtime.h"
#include "hello.h"

__global__ void hello(){

    printf("GPU:: Hello World\n");
}

void show_hello(void){

    hello<<<1,10>>>();
    hipDeviceSynchronize();
    printf("DONE\n");
}
