#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void device_add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void host_add(int *a, int *b, int *c){
    for(int i = 0; i < N; i++){
        c[i] = a[i] + a[i];
    }
}

void print(int *a, int *b, int *c){
    for(int i = 0; i<N; i++){
        printf("\n %d + %d = %d", a[i], b[i], c[i]);
    }
}

void fill(int *data){
    for(int i=0; i<N; i++){
        data[i] = i;
    }
}

int main(){
    int *a, *b, *c;
    int *device_a, *device_b, *device_c;

    int size = N*sizeof(int);

    a = (int *)malloc(size); fill(a);
    b = (int *)malloc(size); fill(b);
    c = (int *)malloc(size),
    // host_add(a, b, c);

    hipMalloc((void *)&device_a, N * sizeof(int));
    hipMalloc((void *)&device_b, N * sizeof(int));
    hipMalloc((void *)&device_c, N * sizeof(int));

    hipMemcpy(device_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    device_add<<<N, 1>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, N*sizeof(int), hipMemcpyDeviceToHost);
    print(a, b, c);

    free(a); free(b); free(c);
    hipFree(device_a); hipFree(device_b); hipFree(device_c);
}