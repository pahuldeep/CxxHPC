﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "iostream"
using namespace std;

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        cout << "Device name: " << prop.name << "\n";
        cout << "Memory Clock Rate (MHz): " << prop.memoryClockRate / 1024 << "\n";
        cout << "Memory Bus Width (bits): " << prop.memoryBusWidth;
        cout << 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6;
        cout << "Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0;
 
        cout << "Shared memory per block(Kbytes) % .1f\n", (float)(prop.sharedMemPerBlock) / 1024.0;
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");

    }
}